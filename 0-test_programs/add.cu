
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

// Kernel function to add the elements of two arrays
__global__
void add(int n, float *x, float *y)
{
    for (int i = 0; i < n; i++)
        y[i] = x[i] + y[i];
}

int main(int argc, char* argv[])
{
    if(argc < 2) 
    {
        std::cerr << "Insert the size\n";
        return 1;
    }

    int N = std::stoi(argv[1]);
    float *h_x, *h_y;

    h_x = new float[N];
    h_y = new float[N];

    // initialize x and y arrays on the host
    for (int i = 0; i < N; i++) {
        h_x[i] = 1.0f;
        h_y[i] = 2.0f;
    }

    float *x, *y; 
    hipMalloc(&x, N*sizeof(float));
    hipMalloc(&y, N*sizeof(float));

    hipMemcpy(x, h_x, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(y, h_y, N*sizeof(float), hipMemcpyHostToDevice);

    // Allocate Unified Memory – accessible from CPU or GPU
    // cudaMallocManaged(&x, N*sizeof(float));
    // cudaMallocManaged(&y, N*sizeof(float));

    // // Run kernel on 1M elements on the GPU
    add<<<1, 1>>>(N, x, y);

    // // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    hipMemcpy(x, h_x, N*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(y, h_y, N*sizeof(float), hipMemcpyDeviceToHost);

    // Check for errors (all values should be 3.0f)
    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
        maxError = fmax(maxError, fabs(h_y[i]-3.0f));
    std::cout << "Max error: " << maxError << std::endl;

    // Free memory
    delete[] h_x;
    delete[] h_y;
    hipFree(x);
    hipFree(y);

    return 0;
}